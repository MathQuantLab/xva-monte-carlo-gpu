/**
 * @file cuda_simulation.cu
 * @author Thomas Roiseux (thomas.roiseux@mathquantlab.com)
 * @brief Implements {@link cuda_simulation.h}
 * @version 1.0
 * @date 2024-04-22
 *
 * @copyright Copyright (c) 2024
 *
 */

#include "../headers/cuda_simulation.h"

#include <hiprand/hiprand_kernel.h>

void CUDA::Simulation::run_simulation(std::map<XVA, double> xva,
                    double m0, double m1,
                    size_t nb_points, double T,
                    std::map<ExternalPaths, std::vector<Vector>> &external_paths,
                    std::map<XVA, Vector> &paths)
{

}