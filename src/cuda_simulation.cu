#include "hip/hip_runtime.h"
/**
 * @file cuda_simulation.cu
 * @author Thomas Roiseux (thomas.roiseux@mathquantlab.com)
 * @brief Implements {@link cuda_simulation.h}
 * @version 1.0
 * @date 2024-04-22
 *
 * @copyright Copyright (c) 2024
 *
 */

#include "../headers/cuda_simulation.h"

#include <hiprand/hiprand_kernel.h>

/**
 * @brief Generate a sample from a Gaussian distribution
 * 
 * @param mean 
 * @param std_dev 
 * @param state 
 * @return double Gaussian sample
 */
__device__ double generate_gaussian_sample(double mean, double std_dev, hiprandState *state)
{
    return mean + std_dev * hiprand_normal_double(state);
}

/**
 * @brief Generate the external path for the interest rate on GPU
 * 
 * @param paths External paths
 * @param m0 Number of paths
 * @param N Size of each path
 * @param T Time horizon
 */
__global__ void generate_external_path_interest_rate(double **paths, size_t *m0, size_t *N, double *T)
{
    double r0 = 0.03;
    double k(0.5);
    double theta = 0.04;
    double sigma = 0.1;

    double dt = *T / *N;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState state;
    hiprand_init(1234, idx, 0, &state);

    if (idx < *m0)
    {
        paths[idx][0] = r0;
        for (size_t i = 1; i < *N; i++)
        {
            double dW = generate_gaussian_sample(0, sqrt(dt), &state);
            paths[idx][i] = paths[idx][i - 1] + k * (theta - paths[idx][i - 1]) * dt + sigma * dW * sqrt(paths[idx][i - 1]);
        }
    }
}

/**
 * @brief Generate the external path for the FX rate on GPU
 * 
 * @param paths External paths
 * @param m0 Number of paths
 * @param N Size of each path
 * @param T Time horizon
 */
__global__ void generate_external_path_fx(double **paths, size_t *m0, size_t *N, double *T)
{
    double S0 = 1.15;
    double mu = 0.02;
    double sigma = 0.1;

    double dt = *T / *N;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState state;
    hiprand_init(1234, idx, 0, &state);

    if (idx < *m0)
    {
        paths[idx][0] = S0;
        for (size_t i = 1; i < *N; i++)
        {
            double dW = generate_gaussian_sample(0, sqrt(dt), &state);
            paths[idx][i] = paths[idx][i - 1] * exp((mu - 0.5 * sigma * sigma) * dt + sigma * dW);
        }
    }
}

__global__ void generate_external_path_equity(double **paths, size_t *m0, size_t *N, double *T)
{
    double S0 = 100;
    double mu = 0.05;
    double sigma = 0.2;

    double dt = *T / *N;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState state;
    hiprand_init(1234, idx, 0, &state);

    if (idx < *m0)
    {
        paths[idx][0] = S0;
        for (size_t i = 1; i < *N; i++)
        {
            double dW = generate_gaussian_sample(0, sqrt(dt), &state);
            paths[idx][i] = paths[idx][i - 1] * exp((mu - 0.5 * sigma * sigma) * dt + sigma * dW);
        }
    }
}

void CUDA::Simulation::run_simulation(const std::map<XVA, double>& xva,
                    size_t m0, size_t m1,
                    size_t nb_points, double T,
                    std::map<ExternalPaths, std::vector<Vector>> &external_paths,
                    std::map<XVA, Vector> &paths)
{
    double *d_T;
    size_t *d_N, *d_m0, *d_m1;
    double **d_paths_interest, **d_paths_fx, **d_paths_equity;
    double **generated_paths = new double*[m0];


    hipMalloc(&d_m0, sizeof(size_t));
    hipMalloc(&d_m1, sizeof(size_t));
    hipMalloc(&d_T, sizeof(double));
    hipMalloc(&d_N, sizeof(size_t));

    hipMemcpy(d_m0, &m0, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_m1, &m1, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_T, &T, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_N, &nb_points, sizeof(size_t), hipMemcpyHostToDevice);

    hipMalloc(&d_paths_interest, m0 * sizeof(double *));

    for (size_t i = 0; i < m0; i++)
    {
        hipMalloc(&d_paths_interest[i], nb_points * sizeof(double));
    }

    generate_external_path_interest_rate<<<m0, 1>>>(d_paths_interest, d_m0, d_N, d_T);
    for (size_t i = 0; i < m0; i++)
    {
        generated_paths[i] = new double[nb_points];
        hipMemcpy(generated_paths[i], d_paths_interest[i], nb_points * sizeof(double), hipMemcpyDeviceToHost);
        external_paths[ExternalPaths::Interest].push_back(Vector(nb_points));
        for (size_t j = 0; j < nb_points; j++)
        {
            external_paths[ExternalPaths::Interest][i][j] = generated_paths[i][j];
        }
    }

    hipMalloc(&d_paths_fx, m0 * sizeof(double *));
    for (size_t i = 0; i < m0; i++)
    {
        hipMalloc(&d_paths_fx[i], nb_points * sizeof(double));
    }

    generate_external_path_fx<<<m0, 1>>>(d_paths_fx, d_m0, d_N, d_T);
    for (size_t i = 0; i < m0; i++)
    {
        generated_paths[i] = new double[nb_points];
        hipMemcpy(generated_paths[i], d_paths_fx[i], nb_points * sizeof(double), hipMemcpyDeviceToHost);
        external_paths[ExternalPaths::FX].push_back(Vector(nb_points));
        for (size_t j = 0; j < nb_points; j++)
        {
            external_paths[ExternalPaths::FX][i][j] = generated_paths[i][j];
        }
    }

    hipMalloc(&d_paths_equity, m0 * sizeof(double *));
    for (size_t i = 0; i < m0; i++)
    {
        hipMalloc(&d_paths_equity[i], nb_points * sizeof(double));
    }

    generate_external_path_equity<<<m0, 1>>>(d_paths_equity, d_m0, d_N, d_T);
    for (size_t i = 0; i < m0; i++)
    {
        generated_paths[i] = new double[nb_points];
        hipMemcpy(generated_paths[i], d_paths_equity[i], nb_points * sizeof(double), hipMemcpyDeviceToHost);
        external_paths[ExternalPaths::Equity].push_back(Vector(nb_points));
        for (size_t j = 0; j < nb_points; j++)
        {
            external_paths[ExternalPaths::Equity][i][j] = generated_paths[i][j];
        }
    }

    for (size_t i = 0; i < m0; i++)
    {
        delete[] generated_paths[i];
    }
    delete[] generated_paths;

    for (size_t i = 0; i < m0; i++)
    {
        hipFree(d_paths_interest[i]);
        hipFree(d_paths_fx[i]);
        hipFree(d_paths_equity[i]);
    }

    hipFree(d_paths_interest);
    hipFree(d_paths_fx);
    hipFree(d_paths_equity);
    hipFree(d_m0);
    hipFree(d_m1);
    hipFree(d_T);
    hipFree(d_N);
}
