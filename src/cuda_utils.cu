/**
 * @file cuda_utils.cu
 * @author Thomas Roiseux (thomas.roiseux@mathquantlab.com)
 * @brief Implments {@link cuda_utils.h}
 * @version 1.0
 * @date 2024-04-19
 *
 * @copyright Copyright (c) 2024
 *
 */

#include "../headers/cuda_utils.h"

bool CUDA::Utils::is_gpu_available() noexcept
{
    int device_count;
    hipGetDeviceCount(&device_count);
    return device_count > 0;
}

void CUDA::Utils::select_gpu(int device_id)
{
    int device_count;
    hipGetDeviceCount(&device_count);

    if (device_id < 0 || device_id >= device_count)
    {
        throw Exception("Invalid device id");
    }

    hipError_t error = hipSetDevice(device_id);

    if (error != hipSuccess)
    {
        throw CUDA::CUDAException("Failed to select device", error);
    }
}

int CUDA::Utils::get_gpu_id()
{
    int device;
    hipGetDevice(&device);
    return device;
}

const char *CUDA::CUDAException::what() const noexcept
{
    std::string error_string = hipGetErrorString(m_error);
    return (m_message + "\n" + error_string).c_str();
}

CUDA::GridBlockSize CUDA::Utils::get_grid_size()
{
    int id = get_gpu_id();

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, id);

    return {prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]};
}

CUDA::GridBlockSize CUDA::Utils::get_grid_size(int device_id)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);

    return {prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]};
}

CUDA::GridBlockSize CUDA::Utils::get_block_size()
{
    int id = get_gpu_id();

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, id);

    return {prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]};
}

CUDA::GridBlockSize CUDA::Utils::get_block_size(int device_id)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);

    return {prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]};
}