/**
 * @file cuda_utils.cu
 * @author Thomas Roiseux (thomas.roiseux@mathquantlab.com)
 * @brief Implments {@link cuda_utils.h}
 * @version 1.0
 * @date 2024-04-19
 * 
 * @copyright Copyright (c) 2024
 * 
 */

#include "../headers/cuda_utils.h"

bool is_gpu_available()
{
    int device_count;
    hipGetDeviceCount(&device_count);
    return device_count > 0;
}