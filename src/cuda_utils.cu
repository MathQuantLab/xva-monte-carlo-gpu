/**
 * @file cuda_utils.cu
 * @author Thomas Roiseux (thomas.roiseux@mathquantlab.com)
 * @brief Implments {@link cuda_utils.h}
 * @version 1.0
 * @date 2024-04-19
 *
 * @copyright Copyright (c) 2024
 *
 */

#include "../headers/cuda_utils.h"

bool CUDA::Utils::is_gpu_available() noexcept
{
    int device_count;
    hipGetDeviceCount(&device_count);
    return device_count > 0;
}

void CUDA::Utils::select_gpu(int device_id)
{
    int device_count;
    hipGetDeviceCount(&device_count);

    if (device_id < 0 || device_id >= device_count)
    {
        throw Exception("Invalid device id");
    }

    hipError_t error = hipSetDevice(device_id);

    if (error != hipSuccess)
    {
        throw CUDA::CUDAException("Failed to select device", error);
    }
}